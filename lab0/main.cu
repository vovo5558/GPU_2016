#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"
#include <ctype.h>
#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}
__global__ void SomeTransform(char *input_gpu, int fsize, int offset) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x + offset * 256;
	if (input_gpu[idx] != '\n') {
		//input_gpu[idx] = '!';
		if(input_gpu[idx]<= 'Z' && input_gpu[idx] >= 'A'){//Big alpha
			input_gpu[idx] += 32;
		}
		else if(input_gpu[idx] <= 'z' && input_gpu[idx] >= 'a'){//Little alpha
			input_gpu[idx] -= 32;
		}
	}
}

int main(int argc, char **argv)
{
	// init, and check
	if (argc != 2) {
		printf("Usage %s <input text file>\n", argv[0]);
		abort();
	}
	FILE *fp = fopen(argv[1], "r");
	if (!fp) {
		printf("Cannot open %s", argv[1]);
		abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// read files
	MemoryBuffer<char> text(fsize+1);
	auto text_smem = text.CreateSync(fsize);
	CHECK;
	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();
	// An example: transform the first 64 characters to '!'
	// Don't transform over the tail
	// And don't transform the line breaks
	int offset = 0;
	while(offset * 256 < fsize){
	SomeTransform<<<4, 64 >>>(input_gpu, fsize, offset);
	offset++;
	}
	puts(text_smem.get_cpu_ro());
	return 0;
}
